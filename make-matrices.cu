
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
using namespace std;

#define main_x 10
#define main_y 100

 void make_matrix(int numRows, int numCols, int *storeHere) {
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            int elemAccess = i * numCols + j;
            storeHere[elemAccess] = rand() % 10;
        }
    }
 };

 void write_matrix(ofstream &file, int *storeHere, int numRows, int numCols) {
    for (size_t i = 0; i < numRows; i++) {
        for (size_t j = 0; j < numCols; j++) {
            file << storeHere[i * numCols + j];
            if (j != numCols - 1) {
                file << ".";
            }
        }
        if (i != numRows - 1) {
            file << "\n";
        }
    }   
}

 int main() {
    int *matrixA, *matrixB;
    size_t matrix_size = main_x * main_y * sizeof(int);

    matrixA = (int *)malloc(matrix_size);
    matrixB = (int *)malloc(matrix_size);

    make_matrix(main_x, main_y, matrixA);
    make_matrix(main_x, main_y, matrixB);

    ofstream output("big-matrices.txt");
    if (!output.is_open()) {
        cerr << "Error: Could not open file for writing.\n";
        return 1;
    }

    write_matrix(output, matrixA, main_x, main_y);
    output << "\n";
    write_matrix(output, matrixB, main_x, main_y);

    output.close();
    
    return 0;
 }